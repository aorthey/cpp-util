#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "cuda_functions.h"
//matrix in shared memory
//extern __shared__ float *matrix;

__device__ void showSharedMemory(float *sM);
__global__ void matrixInverse(float *m);
__global__ void matrixMul(float *matrix, int *thread, int *blockid, int *blockdim);
__global__ void matrixSVD(float *A, float *U, float *S, float *V);
__global__ void getKernelMemory(float *m);
__global__ void showKernelSize(int *m);

// Host function
int
main(int argc, char** argv)
{
	cuda::printCudaDevices();


	//multiplying a MxN and a NxM matrix
	const int Nmults =512;
	const int AH = 512;
	const int AW = 512;
	const int BH = AW;
	const int BW = AH;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	cuda::Matrix A[Nmults];
	cuda::Matrix B[Nmults];
	cuda::Matrix C[Nmults];
	cuda::createRandomMatricesHost(Nmults, AH, AW, BH, BW, A,  B, C);

	printf("Start of %d [%dx%d]*[%dx%d] matrix multiplications....\n",Nmults, AH,AW,BH,BW);
	cuda::performMatrixMultiplicationOnHost(Nmults, A, B, C);
	printf("Done.\n");

	//cuda::printMatrix(A[0]);
	//cuda::printMatrix(B[0]);
	//cuda::printMatrix(C[0]);

	/*** record the time ***/
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime = 0.0f;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("\nelapsed time multiplication on Host: %.4fs\n", elapsedTime/1000);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	/***********************/

	//cuda::printMatrix(C[0]);

	hipEvent_t startDevice, stopDevice;
	hipEventCreate(&startDevice);
	hipEventCreate(&stopDevice);
	hipEventRecord(startDevice, 0);

	//int sizeM = sizeof(cuda::Matrix)*Nmults;

	int sizeA = AH*AW*sizeof(float);
	int sizeB = BH*BW*sizeof(float);
	int sizeC = BW*AH*sizeof(float);

	cuda::Matrix Ad[Nmults];
	cuda::Matrix Bd[Nmults];
	cuda::Matrix Cd[Nmults];
	cuda::Matrix COut[Nmults];

	for(int i=0;i<Nmults;i++){
		Ad[i].width = AW;
		Ad[i].height = AH;
		Bd[i].width = BW;
		Bd[i].height = BH;
		Cd[i].width = BW;
		Cd[i].height = AH;
		hipMalloc((void**)&Ad[i].elements, sizeA);
		hipMalloc((void**)&Bd[i].elements, sizeB);
		hipMalloc((void**)&Cd[i].elements, sizeC);
		hipMemcpy(Ad[i].elements, A[i].elements, sizeA, hipMemcpyHostToDevice);
		hipMemcpy(Bd[i].elements, B[i].elements, sizeB, hipMemcpyHostToDevice);
		hipMemcpy(Cd[i].elements, C[i].elements, sizeC, hipMemcpyHostToDevice);
		COut[i].width = BW;
		COut[i].height = AH;
		COut[i].elements = (float *)malloc(sizeC);
	}

	// set the grid and block sizes
	dim3   DimGrid(1,1,1); //num of blocks (x,y,z)
	dim3   DimBlock(Nmults,1,1); //num of threads
	size_t SharedMemBytes = 1; //bytes of shared memory

	for(int i=0;i<Nmults;i++){
		hipMemcpy(COut[i].elements, Cd[i].elements, sizeC, hipMemcpyDeviceToHost);
	}

	//cuda::printMatrix(COut[0]);

	// invoke the kernel
	printf("\nStart multiplication on Device....\n");
	cuda::performMatrixMultiplicationOnDevice<<< DimGrid, DimBlock, SharedMemBytes >>>(Nmults, Ad, Bd, Cd);
	printf("Done.\n");

	for(int i=0;i<Nmults;i++){
		hipMemcpy(COut[i].elements, Cd[i].elements, sizeC, hipMemcpyDeviceToHost);
	}

	float error = 0.0f;
	for(int i=0;i<Nmults;i++){
		for(int row=0;row<C[i].height;row++){
			float sumUp = 0.0f;
			for(int col=0;col<C[i].width;col++){
				sumUp += abs(*(C[i].elements + row * C[i].width + col)-*(COut[i].elements + row * COut[i].width + col));
			}
			if(sumUp>0.1) printf("matrix no. %d was not correct calculated..\n",i);
			error+=sumUp;
		}
	}
	printf("difference error = %.4f\n",error);
	
	//cuda::printMatrix(COut[0]);

	/*** record the time ***/
	hipEventRecord(stopDevice, 0);
	hipEventSynchronize(stopDevice);
	float elapsedTimeDevice;
	hipEventElapsedTime(&elapsedTimeDevice, startDevice, stopDevice);
	printf("\nelapsed time multiplication on Device (with copy): %.4fs\n", elapsedTimeDevice/1000);
	hipEventDestroy(startDevice);
	hipEventDestroy(stopDevice);
	/***********************/


	/* free everything */
	for(int i=0;i<Nmults;i++){
		hipFree(Ad[i].elements);
		hipFree(Bd[i].elements);
		hipFree(Cd[i].elements);
		free(A[i].elements);
		free(B[i].elements);
		free(C[i].elements);
		free(COut[i].elements);
	}
	return 0;
}

__global__ void
matrixMul(float *matrix, int *thread, int *blockid, int *blockdim)
{
	// determine where in the thread grid we are
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float icl = clock();
	//int idtransx = blockIdx.x * blockDim.x + threadIdx.x;
	__prof_trigger(0);
	matrix[idx] = icl;
	thread[idx] = threadIdx.x;
	blockid[idx] = blockIdx.x;
	blockdim[idx] = gridDim.z;
	__syncthreads();
}


__global__ void
showKernelSize(int *m)
{
	m[0] = gridDim.x;
	m[1] = blockDim.x;
	m[2] = blockIdx.x;
	m[4] = threadIdx.x;

	m[5] = gridDim.y;
	m[6] = blockDim.y;
	m[7] = blockIdx.y;
	m[8] = threadIdx.y;

	m[9] = gridDim.z;
	m[10] = blockDim.z;
	m[11] = blockIdx.z;
	m[12] = threadIdx.z;
}

__device__ void showSharedMemory(float *sM){
	//int idx = blockIdx.x * blockDim.x + threadIdx.x;

}
